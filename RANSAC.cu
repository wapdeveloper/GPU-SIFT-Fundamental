#include "hip/hip_runtime.h"
__global__ void RANSAC_Fundamental(const Point2Df *src, const Point2Df *dst,int pts_num, const int *rand_list, float inlier_threshold, int iterations, int *inliers, float *fundamental)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
	//��������˵��������򷵻�
    if(idx >= iterations) return;   
    int rand_idx[4];    //����б�����
    Point2Df _src[4];   //ԴӰ�����
    Point2Df _dst[4];   //ƥ�����
    //ÿ���̼߳���һ��F����
    float *F = &fundamental[idx*9];
    //��ȡÿ���̸߳��������б�����
    rand_idx[0] = rand_list[idx*4+0];    rand_idx[1] = rand_list[idx*4+1];
    rand_idx[2] = rand_list[idx*4+2];    rand_idx[3] = rand_list[idx*4+3];
	//ȥ���ظ����
    if(rand_idx[0] == rand_idx[1]) return;    if(rand_idx[0] == rand_idx[2]) return;
    if(rand_idx[0] == rand_idx[3]) return;    if(rand_idx[1] == rand_idx[2]) return;
    if(rand_idx[1] == rand_idx[3]) return;    if(rand_idx[2] == rand_idx[3]) return;
    //ͨ������б�������ȡ���������������4�����
    for(int i=0; i < 4; i++) 
	{
        _src[i].x = src[rand_idx[i]].x;
        _src[i].y = src[rand_idx[i]].y;
        _dst[i].x = dst[rand_idx[i]].x;
        _dst[i].y = dst[rand_idx[i]].y;
    }
    //�Ի��������һ�������������³����
    //8�㷨�����������
    int ret = GetFundamental(_src, _dst, F);
    //��������������Χ�����
    inliers[idx] = EvalFundamental(src, dst, pts_num, F, inlier_threshold);
}
